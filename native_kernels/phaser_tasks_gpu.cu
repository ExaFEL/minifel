#include "hip/hip_runtime.h"
/* Copyright 2019 Stanford University
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "hipfft/hipfft.h"

#include "legion.h"

using namespace Legion;

#if 0
__global__
void gpu_phaser_kernel(Rect<3> rect,
                    const FieldAccessor<READ_ONLY, int16_t, 3, coord_t, Realm::AffineAccessor<int16_t, 3, coord_t> > x,
                    unsigned long long *result)
{
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int idy = blockIdx.y * blockDim.y + threadIdx.y;
  const int idz = blockIdx.z * blockDim.z + threadIdx.z;
  const Point<3> p(rect.lo.x + idx, rect.lo.y + idy, rect.lo.z + idz);

  // WARNING: This kernel is really, really inefficient. Please don't
  // use this in any context where performance is important!!!

  // FIXME: CUDA only supports atomicAdd on unsigned. Hopefully this
  // cast does sign extension???
  unsigned long long value = x[p];
  atomicAdd(result, value);
}
#endif

__global__
void phaser_kernel(hipfftComplex *rho_hat, const float *amplitudes)
{
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;

  hipfftComplex rhat = rho_hat[idx];
  float amplitude = amplitudes[idx];

  float phase = atan2(rhat.x, rhat.y);

  // compute the complex exponent:
  // https://docs.scipy.org/doc/numpy/reference/generated/numpy.exp.html
  hipfftComplex exp_phase = { .x = amplitude * cos(phase), .y = amplitude * sin(phase) };

  bool amp_mask = true; // FIXME
  rho_hat[idx] = amp_mask ? exp_phase : rhat;
}

__global__
void ER_update_kernel(hipfftComplex *rho, const hipfftComplex *rho_mod, const bool *support)
{
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;

  rho[idx] = support ? rho_mod[idx] : hipfftComplex { .x = 0, .y = 0 };
}

__global__
void HIO_update_kernel(hipfftComplex *rho, const hipfftComplex *rho_mod, const bool *support, float beta)
{
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;

  hipfftComplex rmod = rho_mod[idx];
  rho[idx] = support ? rmod : hipfftComplex { .x = rho[idx].x - beta * rmod.x, .y = rho[idx].y - beta * rmod.y };
}

void run_phaser_kernel(hipfftComplex *rho_hat, const float *amplitudes, Rect<3> rect)
{
  const dim3 block(256, 1, 1);
  const dim3 grid(rect.volume() / block.x, 1, 1);

  phaser_kernel<<<grid, block>>>(rho_hat, amplitudes);
}

void run_ER_update_kernel(hipfftComplex *rho, const hipfftComplex *rho_mod, const bool *support, Rect<3> rect)
{
  const dim3 block(256, 1, 1);
  const dim3 grid(rect.volume() / block.x, 1, 1);

  ER_update_kernel<<<grid, block>>>(rho, rho_mod, support);
}

void run_HIO_update_kernel(hipfftComplex *rho, const hipfftComplex *rho_mod, const bool *support, Rect<3> rect, float beta)
{
  const dim3 block(256, 1, 1);
  const dim3 grid(rect.volume() / block.x, 1, 1);

  HIO_update_kernel<<<grid, block>>>(rho, rho_mod, support, beta);
}

class FFT {
public:
  FFT(Rect<3> rect, const size_t *strides)
    : rect(rect)
    , strides(strides)
  {
    int n[3] = {int(rect.hi.x - rect.lo.x + 1), int(rect.hi.y - rect.lo.y + 1), int(rect.hi.z - rect.lo.z + 1)};

    if (hipfftPlanMany(&plan, 3, n,
                      NULL, 1, rect.volume(),
                      NULL, 1, rect.volume(),
                      HIPFFT_C2C, 1) != HIPFFT_SUCCESS) {
      assert(false &&"cuFFT error: Plan creation failed");
    }
  }

  ~FFT()
  {
    hipfftDestroy(plan);
  }

  void run(hipfftComplex *input, hipfftComplex *output, int direction)
  {
    if (hipfftExecC2C(plan, input, output, direction) != HIPFFT_SUCCESS) {
      assert(false && "cuFFT error: ExecC2C Forward failed");
    }

    if (hipDeviceSynchronize() != hipSuccess){
      assert(false && "CUDA error: Failed to synchronize");
    }
  }

private:
  Rect<3> rect;
  const size_t *strides;
  hipfftHandle plan;
};

class Phaser {
public:
  Phaser(long er_iter, long hio_iter, double hio_beta,
         const float *amplitudes,
         hipfftComplex *rho, bool *support, Rect<3> rect, const size_t *strides)
    : er_iter(er_iter)
    , hio_iter(hio_iter)
    , hio_beta(hio_beta)
    , amplitudes(amplitudes)
    , rho(rho)
    , support(support)
    , rect(rect)
    , strides(strides)
    , rho_fft(rect, strides)
  {
    hipMalloc((void**)&rho_hat, sizeof(hipfftComplex) * rect.volume());
    if (hipGetLastError() != hipSuccess) {
      assert(false && "CUDA error: Failed to allocate");
    }
  }

  ~Phaser()
  {
    hipFree(rho_hat);
  }

  void run()
  {
    ER_loop();
    HIO_loop();
    ER_loop();
    // shrink_wrap();
  }

private:
  void ER_loop()
  {
    for (long k = 0; k < er_iter; ++k) {
      ER();
    }
  }

  void ER()
  {
    phase();
    run_ER_update_kernel(rho, rho_hat, support, rect);
  }

  void HIO_loop()
  {
    for (long k = 0; k < hio_iter; ++k) {
      HIO();
    }
  }

  void HIO()
  {
    phase();
    run_HIO_update_kernel(rho, rho_hat, support, rect, hio_beta);
  }

  void phase() // updates rho_hat
  {
    rho_fft.run(rho, rho_hat, HIPFFT_FORWARD);
    run_phaser_kernel(rho_hat, amplitudes, rect);
    rho_fft.run(rho_hat, rho_hat, HIPFFT_BACKWARD);
  }

private:
  long er_iter;
  long hio_iter;
  double hio_beta;

  const float *amplitudes;

  hipfftComplex *rho;
  bool *support;
  hipfftComplex *rho_hat;

  Rect<3> rect;
  const size_t *strides;
  FFT rho_fft;
};

struct gpu_phaser_task_args {
  int64_t map[1];
  LogicalRegion diffraction;
  LogicalRegion reconstruction;
  int32_t hio_iter;
  float hio_beta;
  int32_t er_iter;
  FieldID diffraction_fields[3];
  FieldID reconstruction_fields[2];
};

__host__
int64_t gpu_phaser_task(const Task *task,
                        const std::vector<PhysicalRegion> &regions,
                        Context ctx, Runtime *runtime)
{
  assert(task->arglen == sizeof(gpu_phaser_task_args));
  gpu_phaser_task_args args = *(gpu_phaser_task_args *)(task->args);

  assert(regions.size() == 2);

  const FieldAccessor<READ_ONLY, float, 3, coord_t, Realm::AffineAccessor<float, 3, coord_t> > amplitude(regions[0], args.diffraction_fields[2]);
  Rect<3> diffraction_rect = runtime->get_index_space_domain(ctx, regions[0].get_logical_region().get_index_space());
  size_t diffraction_strides[3];
  const float *amplitude_origin = amplitude.ptr(diffraction_rect, diffraction_strides);

  const FieldAccessor<READ_WRITE, bool, 3, coord_t, Realm::AffineAccessor<bool, 3, coord_t> > support(regions[1], args.reconstruction_fields[0]);
  const FieldAccessor<READ_WRITE, hipfftComplex, 3, coord_t, Realm::AffineAccessor<hipfftComplex, 3, coord_t> > rho(regions[1], args.reconstruction_fields[1]);
  Rect<3> rho_rect = runtime->get_index_space_domain(ctx, regions[1].get_logical_region().get_index_space());
  size_t rho_strides[3];
  size_t support_strides[3];
  hipfftComplex *rho_origin = rho.ptr(rho_rect, rho_strides);
  bool *support_origin = support.ptr(rho_rect, support_strides);

  assert(diffraction_rect == rho_rect);
  assert(diffraction_strides[0] == rho_strides[0]);
  assert(diffraction_strides[1] == rho_strides[1]);
  assert(diffraction_strides[2] == rho_strides[2]);
  assert(diffraction_strides[0] == support_strides[0]);
  assert(diffraction_strides[1] == support_strides[1]);
  assert(diffraction_strides[2] == support_strides[2]);

  long hio_iter = args.hio_iter;
  double hio_beta = args.hio_beta;
  long er_iter = args.er_iter;

  Phaser phaser(er_iter, hio_iter, hio_beta,
                amplitude_origin,
                rho_origin, support_origin, rho_rect, rho_strides);
  phaser.run();

#if 0
  const dim3 block(8, 8, 4);
  const dim3 grid(
    ((rect.hi.x - rect.lo.x + 1) + (block.x-1)) / block.x,
    ((rect.hi.y - rect.lo.y + 1) + (block.y-1)) / block.y,
    ((rect.hi.z - rect.lo.z + 1) + (block.z-1)) / block.z);

  unsigned long long result = 0;

  unsigned long long *gpu_result;
  if (hipMalloc(&gpu_result, sizeof(unsigned long long)) != hipSuccess) {
    abort();
  }

  if (hipMemcpy(gpu_result, &result, sizeof(unsigned long long), hipMemcpyHostToDevice) != hipSuccess) {
    abort();
  }

  gpu_phaser_kernel<<<grid, block>>>(rect, x, gpu_result);

  if (hipMemcpy(&result, gpu_result, sizeof(unsigned long long), hipMemcpyDeviceToHost) != hipSuccess) {
    abort();
  }

  int64_t sum = result;
  // printf("gpu sum is %" PRId64 "\n", sum);
  return sum;
#else
  return 0;
#endif
}
