#include "hip/hip_runtime.h"
/* Copyright 2019 Stanford University
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "hipfft/hipfft.h"

#include "legion.h"

using namespace Legion;

enum FieldIDs {
  FID_RHO = 1,
};

#if 0
__global__
void gpu_phaser_kernel(Rect<3> rect,
                    const FieldAccessor<READ_ONLY, int16_t, 3, coord_t, Realm::AffineAccessor<int16_t, 3, coord_t> > x,
                    unsigned long long *result)
{
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int idy = blockIdx.y * blockDim.y + threadIdx.y;
  const int idz = blockIdx.z * blockDim.z + threadIdx.z;
  const Point<3> p(rect.lo.x + idx, rect.lo.y + idy, rect.lo.z + idz);

  // WARNING: This kernel is really, really inefficient. Please don't
  // use this in any context where performance is important!!!

  // FIXME: CUDA only supports atomicAdd on unsigned. Hopefully this
  // cast does sign extension???
  unsigned long long value = x[p];
  atomicAdd(result, value);
}
#endif

class Phaser {
public:
  __host__ Phaser(long er_iter, long hio_iter, double hio_beta,
                  hipfftComplex *rho, Rect<3> rho_rect, const size_t *rho_strides)
    : er_iter(er_iter)
    , hio_iter(hio_iter)
    , hio_beta(hio_beta)
    , rho(rho)
    , rho_rect(rho_rect)
    , rho_strides(rho_strides)
  {}

  __host__ void run()
  {
    ER_loop();
    HIO_loop();
    ER_loop();
    // shrink_wrap();
  }

private:
  __host__ void ER_loop()
  {
    for (long k = 0; k < er_iter; ++k) {
      ER();
    }
  }

  __host__ void ER()
  {
    phase();
  }

  __host__ void HIO_loop()
  {
    for (long k = 0; k < hio_iter; ++k) {
      HIO();
    }
  }

  __host__ void HIO()
  {
  }

  __host__ void phase()
  {
    hipfftComplex *rho_hat = fft(rho, rho_rect, rho_strides);
  }

  __host__ hipfftComplex *fft(hipfftComplex *data, Rect<3> rect, const size_t *strides)
  {
    hipfftHandle plan;
    int n[3] = {int(rect.hi.x - rect.lo.x + 1), int(rect.hi.y - rect.lo.y + 1), int(rect.hi.z - rect.lo.z + 1)};

    hipfftComplex *result;
    hipMalloc((void**)&result, sizeof(hipfftComplex) * rect.volume());
    if (hipGetLastError() != hipSuccess) {
      assert(false && "CUDA error: Failed to allocate");
    }

    if (hipfftPlanMany(&plan, 3, n,
                      NULL, 1, rect.volume(),
                      NULL, 1, rect.volume(),
                      HIPFFT_C2C, 1) != HIPFFT_SUCCESS) {
      assert(false &&"cuFFT error: Plan creation failed");
    }

    if (hipfftExecC2C(plan, data, data, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
      assert(false && "cuFFT error: ExecC2C Forward failed");
    }

    if (hipDeviceSynchronize() != hipSuccess){
      assert(false && "CUDA error: Failed to synchronize");
    }

    hipfftDestroy(plan);
  }


private:
  long er_iter;
  long hio_iter;
  double hio_beta;

  hipfftComplex *rho;
  Rect<3> rho_rect;
  const size_t *rho_strides;
};

__host__
void ER_loop(long er_iter)
{
  for (long k = 0; k < er_iter; ++k) {
  }
}

__host__
int64_t gpu_phaser_task(const Task *task,
                        const std::vector<PhysicalRegion> &regions,
                        Context ctx, Runtime *runtime)
{
  assert(regions.size() == 1);

  const FieldAccessor<READ_WRITE, hipfftComplex, 3, coord_t, Realm::AffineAccessor<hipfftComplex, 3, coord_t> > rho(regions[0], FID_RHO);
  Rect<3> rho_rect = runtime->get_index_space_domain(ctx, regions[0].get_logical_region().get_index_space());
  size_t rho_strides[3];
  hipfftComplex *rho_origin = rho.ptr(rho_rect, rho_strides);

  long hio_iter = 100;
  double hio_beta = 0.1;
  long er_iter = hio_iter / 2;

  Phaser phaser(er_iter, hio_iter, hio_beta, rho_origin, rho_rect, rho_strides);

#if 0
  const dim3 block(8, 8, 4);
  const dim3 grid(
    ((rect.hi.x - rect.lo.x + 1) + (block.x-1)) / block.x,
    ((rect.hi.y - rect.lo.y + 1) + (block.y-1)) / block.y,
    ((rect.hi.z - rect.lo.z + 1) + (block.z-1)) / block.z);

  unsigned long long result = 0;

  unsigned long long *gpu_result;
  if (hipMalloc(&gpu_result, sizeof(unsigned long long)) != hipSuccess) {
    abort();
  }

  if (hipMemcpy(gpu_result, &result, sizeof(unsigned long long), hipMemcpyHostToDevice) != hipSuccess) {
    abort();
  }

  gpu_phaser_kernel<<<grid, block>>>(rect, x, gpu_result);

  if (hipMemcpy(&result, gpu_result, sizeof(unsigned long long), hipMemcpyDeviceToHost) != hipSuccess) {
    abort();
  }

  int64_t sum = result;
  // printf("gpu sum is %" PRId64 "\n", sum);
  return sum;
#else
  return 0;
#endif
}
