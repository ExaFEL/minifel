#include "hip/hip_runtime.h"
/* Copyright 2019 Stanford University
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "legion.h"

#include "native_kernels.h"

#include <stdint.h>
#include <inttypes.h>

using namespace Legion;


__global__
void gpu_sum_kernel(Rect<3> rect,
                    const FieldAccessor<READ_ONLY, int16_t, 3, coord_t, Realm::AffineAccessor<int16_t, 3, coord_t> > x,
                    unsigned long long *result)
{
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int idy = blockIdx.y * blockDim.y + threadIdx.y;
  const int idz = blockIdx.z * blockDim.z + threadIdx.z;
  const Point<3> p(rect.lo.x + idx, rect.lo.y + idy, rect.lo.z + idz);

  // WARNING: This kernel is really, really inefficient. Please don't
  // use this in any context where performance is important!!!

  // FIXME: CUDA only supports atomicAdd on unsigned. Hopefully this
  // cast does sign extension???
  unsigned long long value = x[p];
  atomicAdd(result, value);
}

__host__
int64_t gpu_sum_task(const Task *task,
                     const std::vector<PhysicalRegion> &regions,
                     Context ctx, Runtime *runtime)
{
  assert(regions.size() == 1);

  const FieldAccessor<READ_ONLY, int16_t, 3, coord_t, Realm::AffineAccessor<int16_t, 3, coord_t> > x(regions[0], X_FIELD_ID);

  Rect<3> rect = runtime->get_index_space_domain(ctx,
                  regions[0].get_logical_region().get_index_space());

  const dim3 block(8, 8, 4);
  const dim3 grid(
    ((rect.hi.x - rect.lo.x + 1) + (block.x-1)) / block.x,
    ((rect.hi.y - rect.lo.y + 1) + (block.y-1)) / block.y,
    ((rect.hi.z - rect.lo.z + 1) + (block.z-1)) / block.z);

  unsigned long long result = 0;

  unsigned long long *gpu_result;
  if (hipMalloc(&gpu_result, sizeof(unsigned long long)) != hipSuccess) {
    abort();
  }

  if (hipMemcpy(gpu_result, &result, sizeof(unsigned long long), hipMemcpyHostToDevice) != hipSuccess) {
    abort();
  }

  gpu_sum_kernel<<<grid, block>>>(rect, x, gpu_result);

  if (hipMemcpy(&result, gpu_result, sizeof(unsigned long long), hipMemcpyDeviceToHost) != hipSuccess) {
    abort();
  }

  int64_t sum = result;
  // printf("gpu sum is %" PRId64 "\n", sum);
  return sum;
}
